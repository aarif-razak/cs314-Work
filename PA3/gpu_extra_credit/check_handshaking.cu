/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	
	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i = 0;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/

	for(i = tid; i < numNodes; i += numThreads)
	{
		if(i == strongNeighbor[strongNeighbor[i]]){
			matches[i] = strongNeighbor[i];
		}else{
			matches[i] = -1;
		}

	}
}
