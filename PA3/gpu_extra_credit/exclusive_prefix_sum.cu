/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i = 0;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/

	for(i = tid; i <= numElements; i += numThreads)
	{
		//since this is an exclusive prefix sum, if the distance is 0, every element in the output should be set to the previous element
		//of the intput
		if(distance == 0 ){
			//check for an out of bounds to start
			if( i == 0){
				newSum[i] = 0;
			}else{
				//make everything in the new output equal to the prev of the input
				newSum[i] = oldSum[i-1];
			}
		}else{ //distance/stride != 0, we start adding.
			if(i >= distance){ //first make sure we dont array indexes less than 0.
				newSum[i] = oldSum[i] + oldSum[i-distance]; // the actual scan

			}else{ // if the distance is somehow less than 0
				newSum[i] = oldSum[i];
			}
		}



	}

}
