/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i = 0;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/
	for(i = tid; i < numEdges; i += numThreads)
	{
	if(matches[src[i]] == -1){//check inside the src to be sure
		if(matches[dst[i]] == -1){
			//now we can establish than edge should be ketp
			keepEdges[i] = 1;
		}else{
			keepEdges[i] = 0;
		}
	}else{
		keepEdges[i] = 0;
	} 


	}

}
