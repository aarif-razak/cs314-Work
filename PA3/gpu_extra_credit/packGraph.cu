/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void packGraph_gpu(int * newSrc, int * oldSrc, int * newDst, int * oldDst, int * newWeight, int * oldWeight, int * edgeMap, int numEdges) {
	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i = 0;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/

	for(i = tid; i < numEdges; i += numThreads)
	{

		//2 cases, keeping an edge or not
		/// to check if we keep
		if(edgeMap[i+1] != edgeMap[i]){
			newSrc[edgeMap[i]] = oldSrc[i];
			newDst[edgeMap[i]] = oldDst[i];
			newWeight[edgeMap[i]] = oldWeight[i];
		}



	}
}
