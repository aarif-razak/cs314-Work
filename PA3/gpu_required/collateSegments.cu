/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	
	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/


	for(i = tid; i < numEdges; i += numThreads)
	{

		if(src[i] != src[i+1]){ //we see that the data next to it isnt the same segment, so we've reached the last node
				//then we just update the output array with the scanResults from strongestNeighbor
			output[src[i]] = scanResult[i];
		}


	}

}
