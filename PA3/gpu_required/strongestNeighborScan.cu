/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	/*numEdges is the number of tasks that need to be completed*/
	/*distance is the stride aka the change between each src/dst/weight node check*/


	int numThreads = blockDim.x * gridDim.x; //total number of threads
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  // global index of the thread
	int i = 0;
	/*this code will automatically loop through the number of threads, as long as you refer to each element in the arrays as [tid]*/

	for(i = tid; i < numEdges; i += numThreads)
	{

			/*quickly ensure that the stride is even valid/in the array*/
	if(i-distance >= 0){


		/*check if everything at i is in the same segment*/
	if(src[i]  == src[i-distance]){

		if (oldWeight[i-distance] > oldWeight[i]){ // check if the weight in the next stride is greater than what we have now
			
			newDst[i] = oldDst[i-distance];
			newWeight[i] = oldWeight[i-distance];
			(*madeChanges) = 1;
		}else if(oldWeight[i-distance] == oldWeight[i]){ //nextDoor weight is equal
					
					/*smaller vertexID should be treated as greater*/
					/*it will be already found by dst[i-distance] */
					newDst[i] = oldDst[i-distance];
					newWeight[i] = oldWeight[i];
				

		}else{//in this case, the left oldWeight is greater than the rightside
			newDst[i] = oldDst[i];
			newWeight[i] = oldWeight[i];
			

		}
		//if nothing else, just return the same weight and dst from before
	}else{

			newDst[i] = oldDst[i];
			newWeight[i] = oldWeight[i];
	}

	


	}else{
		newDst[i] = oldDst[i];
			newWeight[i] = oldWeight[i];
			
			}


}
}